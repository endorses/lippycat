// CUDA kernels for pattern matching

#include <hip/hip_runtime.h>


// Simple pattern matching kernel
// Each thread processes one packet against all patterns
__global__ void patternMatchKernel(
    const char* packets,
    const int* packetOffsets,
    int numPackets,
    const char* patterns,
    const int* patternLengths,
    int numPatterns,
    int* results,
    int* resultCount
) {
    int packetIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (packetIdx >= numPackets) {
        return;
    }

    // Get packet data
    int packetStart = packetOffsets[packetIdx];
    int packetEnd = packetOffsets[packetIdx + 1];
    int packetLen = packetEnd - packetStart;

    // Track pattern offset
    int patternOffset = 0;

    // Search for each pattern
    for (int patternIdx = 0; patternIdx < numPatterns; patternIdx++) {
        int patternLen = patternLengths[patternIdx];

        // Skip if pattern longer than packet
        if (patternLen > packetLen) {
            patternOffset += patternLen;
            continue;
        }

        // Boyer-Moore-Horspool style search
        for (int i = 0; i <= packetLen - patternLen; i++) {
            bool match = true;

            // Check if pattern matches at position i
            for (int j = 0; j < patternLen; j++) {
                if (packets[packetStart + i + j] != patterns[patternOffset + j]) {
                    match = false;
                    break;
                }
            }

            if (match) {
                // Record match atomically
                int resultIdx = atomicAdd(resultCount, 1);

                // Store result: [packetIdx, patternIdx, offset, length]
                if (resultIdx < 10000) {  // Safety limit
                    results[resultIdx * 4 + 0] = packetIdx;
                    results[resultIdx * 4 + 1] = patternIdx;
                    results[resultIdx * 4 + 2] = i;
                    results[resultIdx * 4 + 3] = patternLen;
                }

                break;  // Found match, move to next pattern
            }
        }

        patternOffset += patternLen;
    }
}

// Optimized pattern matching using shared memory
__global__ void patternMatchKernelOptimized(
    const char* packets,
    const int* packetOffsets,
    int numPackets,
    const char* patterns,
    const int* patternLengths,
    int numPatterns,
    int* results,
    int* resultCount
) {
    // Shared memory for pattern data (faster access)
    __shared__ char sharedPatterns[1024];
    __shared__ int sharedLengths[32];

    int packetIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // Cooperatively load patterns into shared memory
    if (threadIdx.x == 0) {
        int totalPatternSize = 0;
        for (int i = 0; i < numPatterns && i < 32; i++) {
            sharedLengths[i] = patternLengths[i];
            totalPatternSize += patternLengths[i];
        }

        if (totalPatternSize < 1024) {
            for (int i = 0; i < totalPatternSize; i++) {
                sharedPatterns[i] = patterns[i];
            }
        }
    }

    __syncthreads();

    if (packetIdx >= numPackets) {
        return;
    }

    // Process packet (similar to basic kernel but using shared memory)
    int packetStart = packetOffsets[packetIdx];
    int packetEnd = packetOffsets[packetIdx + 1];
    int packetLen = packetEnd - packetStart;

    int patternOffset = 0;

    for (int patternIdx = 0; patternIdx < numPatterns && patternIdx < 32; patternIdx++) {
        int patternLen = sharedLengths[patternIdx];

        if (patternLen > packetLen) {
            patternOffset += patternLen;
            continue;
        }

        // Search using shared memory patterns
        for (int i = 0; i <= packetLen - patternLen; i++) {
            bool match = true;

            for (int j = 0; j < patternLen; j++) {
                if (packets[packetStart + i + j] != sharedPatterns[patternOffset + j]) {
                    match = false;
                    break;
                }
            }

            if (match) {
                int resultIdx = atomicAdd(resultCount, 1);

                if (resultIdx < 10000) {
                    results[resultIdx * 4 + 0] = packetIdx;
                    results[resultIdx * 4 + 1] = patternIdx;
                    results[resultIdx * 4 + 2] = i;
                    results[resultIdx * 4 + 3] = patternLen;
                }

                break;
            }
        }

        patternOffset += patternLen;
    }
}

// Call-ID extraction kernel (SIP-specific)
__global__ void extractCallIDKernel(
    const char* packets,
    const int* packetOffsets,
    int numPackets,
    char* callIDs,
    int* callIDOffsets,
    int* callIDCount
) {
    int packetIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (packetIdx >= numPackets) {
        return;
    }

    int packetStart = packetOffsets[packetIdx];
    int packetEnd = packetOffsets[packetIdx + 1];
    int packetLen = packetEnd - packetStart;

    // Search for "Call-ID:" or "i:"
    const char* callIDHeader = "Call-ID:";
    const char* shortForm = "\ni:";

    int headerLen = 8;
    int foundOffset = -1;

    // Search for Call-ID:
    for (int i = 0; i <= packetLen - headerLen; i++) {
        bool match = true;
        for (int j = 0; j < headerLen; j++) {
            if (packets[packetStart + i + j] != callIDHeader[j]) {
                match = false;
                break;
            }
        }

        if (match) {
            foundOffset = i + headerLen;
            break;
        }
    }

    // If not found, try short form
    if (foundOffset == -1) {
        headerLen = 3;
        for (int i = 0; i <= packetLen - headerLen; i++) {
            bool match = true;
            for (int j = 0; j < headerLen; j++) {
                if (packets[packetStart + i + j] != shortForm[j]) {
                    match = false;
                    break;
                }
            }

            if (match) {
                foundOffset = i + headerLen;
                break;
            }
        }
    }

    if (foundOffset != -1) {
        // Skip whitespace
        while (foundOffset < packetLen &&
               (packets[packetStart + foundOffset] == ' ' ||
                packets[packetStart + foundOffset] == '\t')) {
            foundOffset++;
        }

        // Find end of Call-ID (until \r or \n)
        int callIDStart = foundOffset;
        int callIDEnd = callIDStart;

        while (callIDEnd < packetLen &&
               packets[packetStart + callIDEnd] != '\r' &&
               packets[packetStart + callIDEnd] != '\n') {
            callIDEnd++;
        }

        int callIDLen = callIDEnd - callIDStart;

        if (callIDLen > 0 && callIDLen < 128) {
            // Record Call-ID
            int resultIdx = atomicAdd(callIDCount, 1);

            if (resultIdx < 10000) {
                int outputOffset = resultIdx * 128;
                callIDOffsets[resultIdx] = outputOffset;
                callIDOffsets[resultIdx + 1] = outputOffset + callIDLen;

                // Copy Call-ID
                for (int i = 0; i < callIDLen; i++) {
                    callIDs[outputOffset + i] = packets[packetStart + callIDStart + i];
                }
            }
        }
    }
}

// C wrapper functions for CGo

extern "C" {

void launchPatternMatchKernel(
    const char* d_packets,
    const int* d_packetOffsets,
    int numPackets,
    const char* d_patterns,
    const int* d_patternLengths,
    int numPatterns,
    int* d_results,
    int* d_resultCount,
    hipStream_t stream
) {
    // Reset result count
    hipMemsetAsync(d_resultCount, 0, sizeof(int), stream);

    // Launch kernel with optimal block size
    int blockSize = 256;
    int numBlocks = (numPackets + blockSize - 1) / blockSize;

    // Use optimized kernel if patterns fit in shared memory
    bool useOptimized = true;
    int totalPatternSize = 0;
    for (int i = 0; i < numPatterns && i < 32; i++) {
        totalPatternSize += d_patternLengths[i];
    }

    if (totalPatternSize > 1024 || numPatterns > 32) {
        useOptimized = false;
    }

    if (useOptimized) {
        patternMatchKernelOptimized<<<numBlocks, blockSize, 0, stream>>>(
            d_packets, d_packetOffsets, numPackets,
            d_patterns, d_patternLengths, numPatterns,
            d_results, d_resultCount
        );
    } else {
        patternMatchKernel<<<numBlocks, blockSize, 0, stream>>>(
            d_packets, d_packetOffsets, numPackets,
            d_patterns, d_patternLengths, numPatterns,
            d_results, d_resultCount
        );
    }
}

void launchCallIDExtractionKernel(
    const char* d_packets,
    const int* d_packetOffsets,
    int numPackets,
    char* d_callIDs,
    int* d_callIDOffsets,
    int* d_callIDCount,
    hipStream_t stream
) {
    // Reset result count
    hipMemsetAsync(d_callIDCount, 0, sizeof(int), stream);

    int blockSize = 256;
    int numBlocks = (numPackets + blockSize - 1) / blockSize;

    extractCallIDKernel<<<numBlocks, blockSize, 0, stream>>>(
        d_packets, d_packetOffsets, numPackets,
        d_callIDs, d_callIDOffsets, d_callIDCount
    );
}

}  // extern "C"